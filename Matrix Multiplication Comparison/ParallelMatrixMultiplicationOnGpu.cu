#include "hip/hip_runtime.h"
%%writefile matrixMul.cu

#include<iostream>
#include<cstdlib>
#include<math.h>
#include<iomanip>
#include<chrono>
#include<hip/hip_runtime.h>
#define rows 5
#define cols 5
using namespace std;

__global__ void gpuMultiplication(int *arr1, int *arr2, int *arr3)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < rows * cols)
    {
        arr3[tid] = arr1[tid] * arr2[tid];
    }
}

int main()
{
    int *harr1, *harr2, *harr3, *darr1, *darr2, *darr3;
    size_t size = rows * cols * sizeof(int);

    harr1 = (int*)malloc(size);
    harr2 = (int*)malloc(size);
    harr3 = (int*)malloc(size);

    hipMalloc(&darr1, size);
    hipMalloc(&darr2, size);
    hipMalloc(&darr3, size);

    for(int i = 0; i < rows; i++)
    {
        for(int j = 0; j <cols; j++)
        {
            harr1[i * cols + j] = rand() % 5;
            harr2[i * cols + j] = rand() % 5;
        }
    }

    hipMemcpy(darr1, harr1, size, hipMemcpyHostToDevice);
    hipMemcpy(darr2, harr2, size, hipMemcpyHostToDevice);

    int threadPerBlock = 256;
    int blocks = (rows * cols + threadPerBlock - 1) / threadPerBlock;

    auto start = chrono::high_resolution_clock::now();

    gpuMultiplication<<<blocks, threadPerBlock>>>(darr1, darr2, darr3);
    hipDeviceSynchronize();

    auto end = chrono::high_resolution_clock::now();
    chrono::duration<double, milli> duration = end - start;

    hipMemcpy(harr3, darr3, size, hipMemcpyDeviceToHost);

    cout<<"Time taken by GPU with parallel execution is "<<duration.count()<<" milliseconds."<<endl;

    hipFree(darr1);
    hipFree(darr2);
    hipFree(darr3);
    free(harr1);
    free(harr2);
    free(harr3);

    return 0;
}